#include "hip/hip_runtime.h"
#include "parallel.cuh"



struct xd
{
	int xd1 = 1;
	int xd2 = 2;
	float xd3 = 3;

};

__global__ void NextDay()
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < nAgents)
	{
			
	}	
}

__host__ void GetDeviceParameters(uint& BlockNum, uint& BlockSize)
{
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	BlockSize = deviceProp.maxThreadsPerBlock;
	BlockNum = ceil(float(nAgents) / float(BlockSize));
	if (BlockSize > nAgents)
		BlockSize = nAgents;
}

__host__ void InitAgents(Agent* &agents)
{
	for (int i = 0; i < nAgents; i++)
	{
		agents[i].deathProb = floatRand(0, maxDeathProb);
		agents[i].extrovertizm = floatRand(0, maxExtravertizmParameter);
		agents[i].getInfectedProb = floatRand(0, maxGetInfectedValue);
		agents[i].infectProb = floatRand(0, maxInfectProb);
	}
}

__host__ void InitPlaces(Place* &places)
{
	for (int i = 0; i < nPlaces; i++)
	{
		places[i].cap = avrCapacity + intRand(-standardDeviation, standardDeviation);
		places[i].contactFactor = floatRand(0, 1);
	}
}

__host__ void InitDisease(Disease* &disease)
{
		disease[0].contagiousness = Dcontagiousness;
		disease[0].duration = Dduration;
}

int main()
{
	srand(time(NULL));

	// Get device parameters to send data asynchronously and specify number of blocks and threads for each block
	int device = hipGetDevice(&device);
	uint BlockNum = 0;
	uint BlockSize = 0;
	GetDeviceParameters(BlockNum, BlockSize);
	printf("BlockNum, BlockSize: %d, %d\n", BlockNum, BlockSize);


	// Malloc memory for arrays with information about infected, healthy and convalescent agents number (Outputs)
	uint* infected;
	uint* healthy;
	uint* convalescent;
	size_t OutputSize = sizeof(uint) * simTime;
	hipMallocManaged(&infected, OutputSize);
	hipMallocManaged(&healthy, OutputSize);
	hipMallocManaged(&convalescent, OutputSize);
	// Make Prefetchs for outputs
	hipMemPrefetchAsync(infected, OutputSize, device, NULL); // ptr, size_t, device, stream
	hipMemPrefetchAsync(healthy, OutputSize, device, NULL);
	hipMemPrefetchAsync(convalescent, OutputSize, device, NULL);
	

	// Allocate agents, places and disease in unified memory
	Agent* agents;
	Disease* disease;
	Place* places;
	size_t AgentSize = sizeof(Agent) * nAgents;
	size_t DiseaseSize = sizeof(Disease);
	size_t PlacesSize = sizeof(Place) * nPlaces;
	hipMallocManaged(&agents, AgentSize);
	hipMallocManaged(&disease, DiseaseSize);
	hipMallocManaged(&places, PlacesSize);
	

	// Memory hints 
	hipMemAdvise(agents, AgentSize, hipMemAdviseSetPreferredLocation, hipCpuDeviceId); // Start on CPU
	hipMemAdvise(disease, DiseaseSize, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
	hipMemAdvise(places, PlacesSize, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
	InitAgents(agents);
	InitPlaces(places);
	InitDisease(disease);
	// Prefetch agents to gpu 
	hipMemPrefetchAsync(agents, AgentSize, device, NULL);
	hipMemPrefetchAsync(disease, DiseaseSize, device, NULL);
	hipMemPrefetchAsync(places, PlacesSize, device, NULL);

	for (int i = 0; i < simTime; i++)
	{
		NextDay << <BlockNum, BlockSize >> > ();
	}
	hipDeviceSynchronize();

	//Get back the outputs
	hipMemPrefetchAsync(infected, OutputSize, hipCpuDeviceId);
	hipMemPrefetchAsync(healthy, OutputSize, hipCpuDeviceId);
	hipMemPrefetchAsync(convalescent, OutputSize, hipCpuDeviceId);


	return 0;
}